
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>();
    hipDeviceReset();
	 
    return 0;
}
