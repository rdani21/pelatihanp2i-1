#include <iostream>
#include <hip/hip_runtime.h>


// kernel menambahkan vector
__global__ 
void tambahVector(
	const float *cVectorA, 
	const float *cVectorB, 
	float *cVectorC, 
	const int cJumlahElemen)
{
	// cari indeks saya
	int idx_ = 0;
}

// fungsi main untuk panggil kernel
int main(void)
{
	// gunakan GPU ke-1 
	hipSetDevice(0);
	const int kJumlahElemen = 25600;
  	size_t ukuran_vector_bytes_ = kJumlahElemen * sizeof(float);
	std::cout << "[Penjumlahan vector dengan jumlah elemen  " << kJumlahElemen << std::endl;
  	float *h_A_ = (float *)malloc(ukuran_vector_bytes_);
  	float *h_B_ = (float *)malloc(ukuran_vector_bytes_);
  	float *h_C_ = (float *)malloc(ukuran_vector_bytes_);

  	if (h_A_ == NULL || h_B_ == NULL || h_C_ == NULL)
  	{
		std::cerr << "Failed to allocate host vectors!\n";
    		exit(-1);
  	}

  	for (int i = 0; i < kJumlahElemen; ++i)
  	{
		h_A_[i] = rand()/(float)RAND_MAX;
		h_B_[i] = rand()/(float)RAND_MAX;
  	}	

  	float *d_A_ = NULL;
	float *d_B_ = NULL;
	float *d_C_ = NULL;

  	hipMalloc((void **)&d_A_, ukuran_vector_bytes_);
	hipMalloc((void **)&d_B_, ukuran_vector_bytes_);
	hipMalloc((void **)&d_C_, ukuran_vector_bytes_);

	std::cout << "Salin input dari host ke  CUDA device\n";

	
  	hipMemcpy(d_A_, h_A_, ukuran_vector_bytes_, hipMemcpyHostToDevice);
  	hipMemcpy(d_B_, h_B_, ukuran_vector_bytes_, hipMemcpyHostToDevice);

  	int threads_per_block_ = 256;
	int blocks_per_grid_ = 1;

	dim3 block(threads_per_block_, 1, 1);
	dim3 grid(blocks_per_grid_, 1, 1);
	
	std::cout << "Peluncuran kernel Cuda dengan ukuran  " << blocks_per_grid_ << " block  " << threads_per_block_ << " threads\n";
  	
	//tambahVector<<<???, ???>>>(??,??,??,??);
	hipError_t err_ = hipGetLastError();
  	if (err_ != hipSuccess)
  	{
		std::cerr << "Gagal meluncurkan kernel Cuda  (error code " << hipGetErrorString(err_) << ")!\n";
    		exit(-1);
	}

	std::cout << "Salin data dari CUDA device ke  host memory\n";
  	hipMemcpy(h_C_, d_C_, ukuran_vector_bytes_, hipMemcpyDeviceToHost);

 	 // verifikasi nilai
 	for (int i = 0; i < kJumlahElemen; ++i)
  	{
    		if (fabs(h_A_[i] + h_B_[i] - h_C_[i]) > 1e-5)
    		{

			std::cerr << "Verifikasi gagal " << i << "!\n";
      			exit(-1);
    		}
  	}


	std::cout << "Test PASSED\n";

  	hipFree(d_A_);
  	hipFree(d_B_);
	hipFree(d_C_);

  	free(h_A_);
  	free(h_B_);
  	free(h_C_);

  	hipDeviceReset();

	std::cout << "Done\n";
  	return 0;
}
