#include <iostream>
#include <hip/hip_runtime.h>


const size_t GRID_SIZE = 100;
const size_t BLOCK_SIZE = 256;

// kernel menambahkan vector
__global__ 
void dotProduct(
	const float *cVectorA, 
	const float *cVectorB, 
	float *dotProductSebagian, 
	const int cJumlahElemen)
{
    	__shared__ float cache[ BLOCK_SIZE ];
	// caching
    	int idx_ = blockIdx.x * blockDim.x + threadIdx.x;
	
    	__syncthreads(); 

	// gunakan idx_ untuk mentrace ukuran block
	
	//hasil akhir pada cache[0]
	if (threadIdx.x == 0) dotProductSebagian[blockIdx.x] = cache[0];
}

// fungsi main untuk panggil kernel
int main(void)
{
	// gunakan GPU ke-1 
	hipSetDevice(0);
	const int kJumlahElemen = GRID_SIZE * BLOCK_SIZE;
  	size_t ukuran_vector_bytes_ = kJumlahElemen * sizeof(float);
	std::cout << "[Penjumlahan vector dengan jumlah elemen  " << kJumlahElemen << std::endl;
  	
	float *h_A_ = (float *)malloc(ukuran_vector_bytes_);
  	float *h_B_ = (float *)malloc(ukuran_vector_bytes_);
  	float *h_C_ = (float *)malloc(GRID_SIZE * sizeof(float));

  	if (h_A_ == NULL || h_B_ == NULL || h_C_ == NULL)
  	{
		std::cerr << "Failed to allocate host vectors!\n";
    		exit(-1);
  	}

	srand(time(NULL));
  	for (int i = 0; i < kJumlahElemen; ++i)
  	{
		h_A_[i] = rand()/(float)RAND_MAX;
		h_B_[i] = rand()/(float)RAND_MAX;
  	}	

  	float *d_A_ = NULL;
	float *d_B_ = NULL;
	float *d_C_ = NULL;

  	hipMalloc((void **)&d_A_, ukuran_vector_bytes_);
	hipMalloc((void **)&d_B_, ukuran_vector_bytes_);
	hipMalloc((void **)&d_C_, GRID_SIZE * sizeof(float));

	std::cout << "Salin input dari host ke  CUDA device\n";

	
  	hipMemcpy(d_A_, h_A_, ukuran_vector_bytes_, hipMemcpyHostToDevice);
  	hipMemcpy(d_B_, h_B_, ukuran_vector_bytes_, hipMemcpyHostToDevice);


	dim3 block(BLOCK_SIZE, 1, 1);
	dim3 grid(GRID_SIZE, 1, 1);
	
	std::cout << "Peluncuran kernel Cuda dengan ukuran  " << GRID_SIZE << " block  " << BLOCK_SIZE << " threads\n";
  	
	dotProduct<<<grid,block>>>(d_A_,d_B_,d_C_,kJumlahElemen);
	hipError_t err_ = hipGetLastError();
  	if (err_ != hipSuccess)
  	{
		std::cerr << "Gagal meluncurkan kernel Cuda  (error code " << hipGetErrorString(err_) << ")!\n";
    		exit(-1);
	}

	std::cout << "Salin data dari CUDA device ke  host memory\n";
  	hipMemcpy(h_C_, d_C_, GRID_SIZE * sizeof(float), hipMemcpyDeviceToHost);

	float resultGPU  = 0.0;
	for (int i=0;i<GRID_SIZE;i++) resultGPU += h_C_[i];
	
	float resultCPU	 = 0.0;
	for (int i=0;i<kJumlahElemen;i++) resultCPU += h_A_[i] * h_B_[i];

	std::cout << "GPU = " << resultGPU << std::endl;
	std::cout << "CPU = " << resultCPU << std::endl;

	if (fabs(resultGPU - resultCPU) < 1e-1)
		std::cout << "Test PASSED\n";
	else
		std::cout << "Test FAILED\n";


  	hipFree(d_A_);
  	hipFree(d_B_);
	hipFree(d_C_);

  	free(h_A_);
  	free(h_B_);
  	free(h_C_);

  	hipDeviceReset();

	std::cout << "Done\n";
  	return 0;
}
