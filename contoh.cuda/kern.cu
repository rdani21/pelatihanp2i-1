#include <stdio.h>
#include <hip/hip_runtime.h>

// tugas1: alokasi memori dan transfer dari device ke host
#define N 12

__global__ void
kern(int *A)
{
    	int idx = blockDim.x * blockIdx.x + threadIdx.x;
 	A[idx] = idx;
}

/**
 * Host main routine
 */
int   main(void)
{
	// alokasikan memori, dan salin nilainya
	int *A = (int *) malloc (N*sizeof(int)); //alokasi di host

	
	//alokasi global memori di device memakai cuda malloc
	int *dev_A ;
	hipMalloc(&dev_A,N*sizeof(int));

	//copy data dari host ke device
	//cudaMemcpy(dev_A,A,N*sizeof(int),cudaMemcpyHostToDevice);
 
	dim3 grid,block;
	block.x = 4;
	grid.x = 12/block.x;

	kern<<<grid,block>>>(dev_A);
	// copy result

	//copy hasil dari device ke host
        hipMemcpy(A,dev_A,N*sizeof(int),hipMemcpyDeviceToHost);

	for(int i=0;i<N;i++){
		printf("A[%d]=%d\n,",i,A[i]);
	}
	free(A);
	hipFree(dev_A);

	return 0;
}
