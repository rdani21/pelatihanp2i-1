#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

<<<<<<< HEAD
// tugas1: alokasi memori dan transfer dari device ke host
#define N 12
=======
#define N 12
// tugas 1: alokasi memori dan transfer dari device ke host
>>>>>>> b3ed746430a99b81299d328a171dbd11bd1df781

__global__ void kern(int *A)
{
    	int idx = blockDim.x * blockIdx.x + threadIdx.x;
 	A[idx] = idx;
}

/**
 * Host main routine
 */
int   main(void)
{
	// alokasikan memori, dan salin nilainya
<<<<<<< HEAD
	int *A = (int *) malloc (N*sizeof(int)); //alokasi di host

	
	//alokasi global memori di device memakai cuda malloc
	int *dev_A ;
	hipMalloc(&dev_A,N*sizeof(int));

	//copy data dari host ke device
	//hipMemcpy(dev_A,A,N*sizeof(int),hipMemcpyHostToDevice);
 
	dim3 grid,block;
	block.x = 4;
	grid.x = 12/block.x;

	kern<<<grid,block>>>(dev_A);
	// copy result

	//copy hasil dari device ke host
        hipMemcpy(A,dev_A,N*sizeof(int),hipMemcpyDeviceToHost);

	for(int i=0;i<N;i++){
		printf("A[%d]=%d\n,",i,A[i]);
	}
	free(A);
	hipFree(dev_A);

=======
	int *A = (int *) malloc(N*sizeof(int)); // alokasi memory di host
	int *d_A;
	hipMalloc((void **)&d_A,N*sizeof(int)); // alokasi memori di device
	hipMemcpy(d_A,A,N*sizeof(int),hipMemcpyHostToDevice); // 

	dim3 grid,block;
	block.x = 4;
	grid.x = 12/block.x;
	kern<<<grid,block>>>(d_A);
	hipMemcpy(A,d_A,N*sizeof(int),hipMemcpyDeviceToHost); // copy device ke host 
	// copy result
	for (int i = 0;i < N;i++) 
		printf("A[%d] = %d\n",i,A[i]);
	free(A);
	hipFree(d_A);
>>>>>>> b3ed746430a99b81299d328a171dbd11bd1df781
	return 0;
}
